#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <windows.h>
#define BLOCK_SIZE  16          // submatrix size
#define N           4000        // matrix size is N*N

__global__ void matMult ( double * a, double * b, int n, double * c )
{
    int bx = blockIdx.x;        // block index
    int by = blockIdx.y;

    int tx = threadIdx.x;       // thread index
    int ty = threadIdx.y;
    
                                // Index of the first sub-matrix of A processed by the block
    int aBegin = n * BLOCK_SIZE * by;
    int aEnd = aBegin + n - 1;
                                // Step size used to iterate through the sub-matrices of A
    int aStep = BLOCK_SIZE;
                                // Index of the first sub-matrix of B processed by the block
    int bBegin = BLOCK_SIZE * bx;
                                // Step size used to iterate through the sub-matrices of B
    int bStep = BLOCK_SIZE * n;
    double sum = 0;           // computed subelement
    
    for ( int ia = aBegin, ib = bBegin; ia <= aEnd; ia += aStep, ib += bStep )
    {
                            // Shared memory for the sub-matrix of A
        __shared__ float as [BLOCK_SIZE][BLOCK_SIZE];
                            // Shared memory for the sub-matrix of B
        __shared__ float bs [BLOCK_SIZE][BLOCK_SIZE];
        
                            // Load the matrices from global memory to shared memory;
        as [ty][tx] = a [ia + n * ty + tx];
        bs [ty][tx] = b [ib + n * ty + tx];
        
        __syncthreads();    // Synchronize to make sure the matrices are loaded
        
                            // Multiply the two matrices together;
        for ( int k = 0; k < BLOCK_SIZE; k++ )
            sum += as [ty][k] * bs [k][tx];
            
                            // Synchronize to make sure that the preceding
                            // computation is done before loading two new
                            // sub-matrices of A and B in the next iteration
        __syncthreads();
    }
    
                            // Write the block sub-matrix to global memory;
                            // each thread writes one element
    int ic = n * BLOCK_SIZE * by + BLOCK_SIZE * bx;
    
    c [ic + n * ty + tx] = sum;
}

int main ( int argc, char *  argv [] )
{	
    int numBytes = N * N * sizeof ( double );

                    // allocate host memory
    double * a = new double [N*N];
    double * b = new double [N*N];
    double * c = new double [N*N];
    
    for ( int i = 0; i < N; i++ )
        for ( int j = 0; j < N; j++ )
        {
            a [i] = 5*i+j*7+13;
            b [i] = 5*i+j*7+13;
        }
        
                    // allocate device memory
    double * adev = NULL;
    double * bdev = NULL;
    double * cdev = NULL;
    
    hipMalloc ( (void**)&adev, numBytes );
    hipMalloc ( (void**)&bdev, numBytes );
    hipMalloc ( (void**)&cdev, numBytes );

                    // set kernel launch configuration
    dim3 threads ( BLOCK_SIZE, BLOCK_SIZE );
    dim3 blocks  ( N / threads.x, N / threads.y);

                    // create cuda event handles
    hipEvent_t start, stop;
    float gpuTime = 0.0f;

    hipEventCreate ( &start );
    hipEventCreate ( &stop );
    
                    // asynchronously issue work to the GPU (all to stream 0)
    hipEventRecord ( start, 0 );
    hipMemcpy      ( adev, a, numBytes, hipMemcpyHostToDevice );
    hipMemcpy      ( bdev, b, numBytes, hipMemcpyHostToDevice );
    
    matMult<<<blocks, threads>>> ( adev, bdev, N, cdev );
    
    hipMemcpy      ( c, cdev, numBytes, hipMemcpyDeviceToHost );
    hipEventRecord ( stop, 0 );

    hipEventSynchronize ( stop );
    hipEventElapsedTime ( &gpuTime, start, stop );

                        // print the cpu and gpu times
    printf("time spent executing by the GPU: %0.f millseconds\n", gpuTime );

                    // release resources
    hipEventDestroy ( start );
    hipEventDestroy ( stop  );
    hipFree         ( adev  );
    hipFree         ( bdev  );
    hipFree         ( cdev  );

    delete a;
    delete b;
    delete c;
    system("Pause");
    return 0;
}